#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  // I have already implemented the tiled multiplication in mp2 as I thought it was required.
  __shared__ float subTileM[16][16];
  __shared__ float subTileN[16][16];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * 16 + ty;
  int Col = bx * 16 + tx;
  float Pvalue = 0;

  for (int m = 0; m < ceil(numAColumns / 16.0); ++m) {
    if (Row < numARows && m * 16 + tx < numAColumns) {
      subTileM[ty][tx] = A[Row * numAColumns + m * 16 + tx];
    } else {
      subTileM[ty][tx] = 0.;
    }
    if (m * 16 + ty < numBRows && Col < numBColumns) {
      subTileN[ty][tx] = B[(m * 16 + ty) * numBColumns + Col];
    } else {
      subTileN[ty][tx] = 0.;
    }
    __syncthreads();
    for (int k = 0; k < 16; ++k) {
      Pvalue += subTileM[ty][k] * subTileN[k][tx];
    }
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Pvalue;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));

  //@@ Allocate GPU memory here
  float *deviceA;
  float *deviceB;
  float *deviceC;
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  // set tile width to be 16
  dim3 DimGrid(ceil(numCColumns / 16.0), ceil(numCRows / 16.0), 1);
  dim3 DimBlock(16, 16, 1);

  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);


  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  //@@Free the hostC matrix

  return 0;
}
